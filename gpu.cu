
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
using namespace std;

__global__ void MatAdd(float *A, float *B, float *C, int times) {
  int block_idx = blockIdx.x * blockDim.x * blockDim.y;
  int i = threadIdx.x + blockDim.x * threadIdx.y + block_idx;
  for (int k = 0; k < times; k++) C[i] += A[i] * 3.14 + B[i] / 3.14;
}

int main(int argc, char **argv) {
  int N = atoi(argv[1]);
  int times = atoi(argv[2]);
  cout << N << "*" << N << ", " << times << "times" << endl;

  float A[N * N], B[N * N], C[N * N];
  for (int j = 0; j < N; j++) {
    for (int i = 0; i < N; i++) {
      A[i + j * N] = i;
      B[i + j * N] = j;
      C[i + j * N] = 0.0;
    }
  }

  auto start = chrono::system_clock::now();

  float *a, *b, *c;
  hipMalloc(&a, N * N * sizeof(float));
  hipMalloc(&b, N * N * sizeof(float));
  hipMalloc(&c, N * N * sizeof(float));

  hipMemcpy(a, A, N * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b, B, N * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(c, C, N * N * sizeof(float), hipMemcpyHostToDevice);

  MatAdd<<<N, N>>>(a, b, c, times);

  hipMemcpy(C, c, N * N * sizeof(float), hipMemcpyDeviceToHost);
  auto end = chrono::system_clock::now();
  auto dur = end - start;
  cerr << (double)(chrono::duration_cast<chrono::nanoseconds>(dur).count()) /
              1000000
       << endl;

  /*
  cout << "C" << endl;
  for (int j = N - 1; j < N; j++) {
    for (int i = 0; i < N; i++) {
      cout << C[i + j * N] << ' ';
    }
    cout << endl;
  }
  */

  hipFree(a);
  hipFree(b);
  hipFree(c);

  return 0;
}
